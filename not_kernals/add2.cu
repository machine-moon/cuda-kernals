#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA Kernel for adding two numbers
__global__ void add(int *a, int *b, int *c) { *c = *a + *b; }

int main() {
  // Host copies of a, b, c
  int a, b, c;
  // Device copies of a, b, c
  int *d_a, *d_b, *d_c;
  int size = sizeof(int);

  // Setup input values
  a = 2;
  b = 7;

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU with one thread
  add<<<1, 1>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

  // Cleanup
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  printf("Result of %d + %d = %d\n", a, b, c);

  return 0;
}