#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloFromGPU() {
    printf("Hello World from GPU!\n");
}

int main() {
    helloFromGPU<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
